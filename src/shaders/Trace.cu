#include "hip/hip_runtime.h"
#include "Trace.cuh"
#include "Ray.hpp"
#include "geometry/Sphere.hpp"

#include <optix_device.h>

#define SAMPLES_PER_PIXEL 4
#define MAX_DEPTH 16

OPTIX_BOUNDS_PROGRAM(LambertianSpheres)(const void *geom_data, box3f &prim_bounds, const int prim_id) {
    Geometry::Sphere::bounds<Geometry::LambertianSpheresGeom>(geom_data, prim_bounds, prim_id);
}

OPTIX_INTERSECT_PROGRAM(LambertianSpheres)() {
    Geometry::Sphere::intersect<Geometry::LambertianSpheresGeom>();
}

OPTIX_CLOSEST_HIT_PROGRAM(LambertianSpheres)() {
    Geometry::Sphere::closest_hit<Geometry::LambertianSpheresGeom, Material::Lambertian>();
}

inline __device__
vec3f miss_color(const Ray& ray) {
    const vec3f ray_dir = normalize(ray.direction);
    const float t = 0.5f * (ray_dir.y + 1.0f);
    const vec3f c = (1.0f - t) * vec3f(1.0f) + t * vec3f(0.5f, 0.7f, 1.0f);
    return c;
}

inline __device__
vec3f trace_path(const RayGenData& self, Ray& ray, RayData::Record& prd) {
    vec3f accum_attenuation = 1.f;

    for (int depth = 0; depth < MAX_DEPTH; depth++) {
        traceRay(self.world, ray, prd);

        if (prd.out.scatter_event == RayData::ScatterEvent::RayMissed) {
            // Missed the scene, return background color
            return accum_attenuation * prd.out.attenuation;
        }
        else if (prd.out.scatter_event == RayData::ScatterEvent::RayCancelled) {
            // Hit light source
            return vec3f(0.f);
        }
        else {
            const float pdf = prd.out.pdf;
            const vec3f brdf = prd.out.attenuation;

            const vec3f throughput = brdf / pdf;
            float roulette_weight =
                1.0f - clamp(max(max(throughput.x, throughput.y), throughput.z), 0.0f, 1.0f);
            if (depth <= 3) roulette_weight = 0.f;

            if (prd.random() < roulette_weight) {
                return vec3f(0.f);
            }

            accum_attenuation *= (throughput / (1.0f - roulette_weight));

            ray = Ray(
                prd.out.scattered_origin,
                prd.out.scattered_direction,
                1e-3f,
                1e10f
                );
        }
    }

    return vec3f(0.f);
}

OPTIX_RAYGEN_PROGRAM(RayGen)() {
    const RayGenData& self = owl::getProgramData<RayGenData>();
    // Get our pixel indices
    const vec2i pixel_id = owl::getLaunchIndex();
    const int pboOfs = pixel_id.x + self.pbo_size.x * pixel_id.y;

    // Build primary rays
    RayData::Record prd;
    prd.random.init(pboOfs, self.launch->frame.id);

    vec3f color = 0.f;
    for (int sample_id = 0; sample_id < SAMPLES_PER_PIXEL; sample_id++) {
        // Build primary ray
        Ray ray;

        const vec2f pixel_offset(prd.random(), prd.random());
        const vec2f uv = (vec2f(pixel_id) + pixel_offset) / vec2f(self.pbo_size);
        const vec3f origin = self.launch->camera.pos;
        const vec3f direction = self.launch->camera.dir_00
            + uv.x * self.launch->camera.dir_du
            + uv.y * self.launch->camera.dir_dv;

        ray.origin = origin;
        ray.direction = normalize(direction);

        // Trace
        color += trace_path(self, ray, prd);
    }

    if (self.launch->dirty) {
        self.pbo_ptr[pboOfs] = vec4f(color * (1.f / SAMPLES_PER_PIXEL), 1.f);
    } else {
        self.pbo_ptr[pboOfs] += vec4f(color * (1.f / SAMPLES_PER_PIXEL), 1.f);
    }
}

// OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)() {
//     vec3f& prd = owl::getPRD<vec3f>();
//
//     const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
//
//     // Compute normal
//     const int prim_id = optixGetPrimitiveIndex();
//     const vec3i index = self.index[prim_id];
//     const vec3f v0 = self.vertex[index.x];
//     const vec3f v1 = self.vertex[index.y];
//     const vec3f v2 = self.vertex[index.z];
//     const vec3f n = normalize(cross(v1 - v0, v2 - v0));
//
//     const vec3f ray_dir = optixGetWorldRayDirection();
// }

OPTIX_MISS_PROGRAM(Miss)() {
    const MissProgData &self = owl::getProgramData<MissProgData>();
    vec3f dir = optixGetWorldRayDirection();
    dir = normalize(dir);

    // Convert to spherical coords
    const float theta = atan2f(dir.z, dir.x);
    const float phi = acosf(dir.y);

    // Convert to texture coordinates
    const float u = theta / (2.0f * M_PIf) + 0.5f;
    const float v = phi / M_PIf;

    // Retrieve environment map color
    const vec4f bg_color = tex2D<float4>(self.env_map, u, v);

    RayData::Record& prd = owl::getPRD<RayData::Record>();
    prd.out.scatter_event = RayData::ScatterEvent::RayMissed;
    prd.out.attenuation = vec3f(bg_color.x, bg_color.y, bg_color.z);
}
