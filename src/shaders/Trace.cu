#include "hip/hip_runtime.h"
#include "Trace.cuh"
#include "trace/Ray.hpp"
#include "geometry/Sphere.hpp"

#include <optix_device.h>

#include "geometry/TriangleMesh.hpp"

#define SAMPLES_PER_PIXEL 2
#define MAX_DEPTH 50

/**
 * @brief Y-up
 */
inline __device__
vec3f random_in_cosine_weighted_hemisphere(LCG<8> &rng) {
    const float r1 = rng();
    const float r2 = rng();
    float phi = 2.0f * M_PI * r1;
    const float y = sqrtf(1.0f - r2);
    const float x = cosf(phi) * sqrtf(r2);
    const float z = sinf(phi) * sqrtf(r2);
    return normalize(vec3f(x, y, z));
}

__device__
static bool scatter(const vec3f diffuse,
                    const vec3f &P,
                    vec3f N,
                    Trace::Record &prd
) {
    const vec3f W_o = optixGetWorldRayDirection();

    // Flip
    if (dot(N, W_o) > 0.0f) {
        N = -N;
    }
    N = normalize(N);

    // Create onb
    const vec3f ref = N.y > 0.9999f ? vec3f(1, 0, 0) : vec3f(0, 1, 0);
    const vec3f T = normalize(cross(ref, N));
    const vec3f B = normalize(cross(N, T));

    // T*x + N*y + B*z to convert local (x,y,z) to world
    const vec3f w_i = random_in_cosine_weighted_hemisphere(prd.random);
    const vec3f W_i = w_i.x * T + w_i.y * N + w_i.z * B;

    // Update the record
    // Note we include the cos(theta) term in the attenuation
    prd.out.scattered_origin = P;
    prd.out.scattered_direction = W_i;
    prd.out.attenuation = (diffuse / (M_PIf)) * w_i.y;
    prd.out.normal = N;
    prd.out.pdf = w_i.y / M_PIf;
    return true;
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)() {
    const auto& self = owl::getProgramData<Geometry::TriangleMesh>();
    Trace::Record& prd = owl::getPRD<Trace::Record>();
    // Ray data
    const vec3f ray_org = optixGetWorldRayOrigin();
    const vec3f ray_dir = optixGetWorldRayDirection();
    const float hit_t = optixGetRayTmax();
    const vec3f hit_point = ray_org + hit_t * ray_dir;
    // Tri data
    const int prim_id = optixGetPrimitiveIndex();
    const vec3ui nindex = self.normal_indices[prim_id];
    const vec3f& n0 = self.normals[nindex.x];
    const vec3f& n1 = self.normals[nindex.y];
    const vec3f& n2 = self.normals[nindex.z];
    const vec2f bary = optixGetTriangleBarycentrics();
    const vec3f N = normalize(bary.x * n0 + bary.y * n1 + (1.0f - bary.x - bary.y) * n2);
    // Scatter
    // const vec3ui tindex = self.texcoord_indices[prim_id];
    // const vec2f& t0 = self.tex_coords[tindex.x];
    // const vec2f& t1 = self.tex_coords[tindex.y];
    // const vec2f& t2 = self.tex_coords[tindex.z];
    // vec2f uv = bary.x * t0 + bary.y * t1 + (1.0f - bary.x - bary.y) * t2;
    // uv.x = fmodf(uv.x, 1.0f);
    // uv.y = fmodf(uv.y, 1.0f);
    scatter({0.8, 0.8, 0.8}, hit_point, N, prd);
    prd.out.scatter_event = Trace::ScatterEvent::RayScattered;
}

OPTIX_BOUNDS_PROGRAM(LambertianSpheres)(const void *geom_data, box3f &prim_bounds, const int prim_id) {
    Geometry::Sphere::bounds<Geometry::LambertianSpheresGeom>(geom_data, prim_bounds, prim_id);
}

OPTIX_INTERSECT_PROGRAM(LambertianSpheres)() {
    Geometry::Sphere::intersect<Geometry::LambertianSpheresGeom>();
}

OPTIX_CLOSEST_HIT_PROGRAM(LambertianSpheres)() {
    Geometry::Sphere::closest_hit<Geometry::LambertianSpheresGeom, Material::Lambertian>();
}

inline __device__
vec3f miss_color(const Ray& ray) {
    const vec3f ray_dir = normalize(ray.direction);
    const float t = 0.5f * (ray_dir.y + 1.0f);
    const vec3f c = (1.0f - t) * vec3f(1.0f) + t * vec3f(0.5f, 0.7f, 1.0f);
    return c;
}

// Currently just a cosine-weighted hemisphere
inline __device__
float bsdf_pdf(const vec3f& W_i, const vec3f& N) {
    return max(0.0f, dot(W_i, N) / M_PIf);
}

inline __device__
int sample_env_discrete(const RayGenData& self, Trace::Record& prd) {
    const int size = self.env.size.x * self.env.size.y;
    int idx = prd.random() * size;
    idx = min(idx, size - 1);

    if (prd.random() < self.env.alias_pdf[idx]) {
        return idx;
    } else {
        const int alias = self.env.alias_i[idx];
        return alias;
    }
}

inline __device__
vec3f sample_env_ray(const RayGenData& self, Trace::Record& prd) {
    int idx = sample_env_discrete(self, prd);

    int y = idx / self.env.size.x;
    int x = idx % self.env.size.x;

    // Convert to 2d texture coordinates
    const float u = (x + 0.5) / self.env.size.x;
    const float v = (y + 0.5) / self.env.size.y;

    // Convert to spherical
    const float theta = 2.0f * M_PIf * (u - 0.5f);
    const float phi = M_PIf * v;

    // Convert to cartesian
    const float ray_x = cosf(theta) * sinf(phi);
    const float ray_y = cosf(phi);
    const float ray_z = sinf(theta) * sinf(phi);

    return vec3f(ray_x, ray_y, ray_z);
}

inline __device__
float get_env_pdf(const RayGenData& self, const vec3f& dir, Trace::Record& prd) {
    // Convert to spherical coords
    const float theta = atan2f(dir.z, dir.x);
    const float phi = acosf(dir.y);

    // Convert to texture coordinates
    const float u = theta / (2.0f * M_PIf) + 0.5f;
    const float v = phi / M_PIf;

    // Convert to index in pdf table
    const int x = static_cast<int>(u * self.env.size.x);
    const int y = static_cast<int>(v * self.env.size.y);
    int idx = y * self.env.size.x + x;
    idx = min(idx, self.env.size.x * self.env.size.y - 1);

    // Get pdf
    return self.env.pdf[idx];
}

inline __device__
bool traceShadowRay(vec3f origin, vec3f direction, Trace::Record& prd) {
    const float tmin = 1e-3f;
    const float tmax = 1e10f;
    Ray ray(origin, direction, tmin, tmax);
    prd.out.scatter_event = Trace::ScatterEvent::RayMissed;
    traceRay(owl::getProgramData<RayGenData>().world, ray, prd);
    // True if missed
    return prd.out.scatter_event != Trace::ScatterEvent::RayScattered;
}

inline __device__
vec3f trace_path(const RayGenData& self, Ray& ray, Trace::Record& prd) {
    vec3f accum_attenuation = 1.f;

    for (int depth = 0; depth < MAX_DEPTH; depth++) {
        traceRay(self.world, ray, prd);

        // BG
        if (prd.out.scatter_event == Trace::ScatterEvent::RayMissed) {
            // Missed the scene, return background color
            return accum_attenuation * prd.out.attenuation;
        }

        // Light (not implemented)
        if (prd.out.scatter_event == Trace::ScatterEvent::RayCancelled) {
            return vec3f(0.f);
        }

        const vec3f brdf = prd.out.attenuation;
        vec3f dir = prd.out.scattered_direction;
        const float pdf = bsdf_pdf(dir, prd.out.normal);

        const vec3f throughput = brdf / pdf;
        float roulette_weight =
            1.0f - clamp(max(max(throughput.x, throughput.y), throughput.z), 0.3f, 1.0f);
        if (depth <= 3) roulette_weight = 0.f;

        if (prd.random() < roulette_weight) {
            return vec3f(0.f);
        }

        vec3f l = (throughput / (1.0f - roulette_weight));
        accum_attenuation *= l;

        ray = Ray(
            prd.out.scattered_origin,
            dir,
            1e-3f,
            1e10f
        );
    }

    return vec3f(0.f);
}

OPTIX_RAYGEN_PROGRAM(RayGen)() {
    const RayGenData& self = owl::getProgramData<RayGenData>();
    // Get our pixel indices
    const vec2i pixel_id = owl::getLaunchIndex();
    const int pboOfs = pixel_id.x + self.pbo_size.x * pixel_id.y;

    // Build primary rays
    Trace::Record prd;
    prd.random.init(pboOfs, self.launch->frame.id);

    vec3f color = 0.f;
    for (int sample_id = 0; sample_id < SAMPLES_PER_PIXEL; sample_id++) {
        // Build primary ray
        Ray ray;

        const vec2f pixel_offset(prd.random(), prd.random());
        const vec2f uv = (vec2f(pixel_id) + pixel_offset) / vec2f(self.pbo_size);
        const vec3f origin = self.launch->camera.pos;
        const vec3f direction = self.launch->camera.dir_00
            + uv.x * self.launch->camera.dir_du
            + uv.y * self.launch->camera.dir_dv;

        ray.origin = origin;
        ray.direction = normalize(direction);

        // Trace
        prd.out.pdf = 1.f;
        color += trace_path(self, ray, prd);
    }

    if (isnan(color.x) || isnan(color.y) || isnan(color.z)) {
        color = vec3f(0.f);
    }

    if (isinf(color.x) || isinf(color.y) || isinf(color.z)) {
        color = vec3f(0.f);
    }

    if (self.launch->dirty) {
        self.pbo_ptr[pboOfs] = vec4f(color * (1.f / SAMPLES_PER_PIXEL), 1.f);
    } else {
        self.pbo_ptr[pboOfs] += vec4f(color * (1.f / SAMPLES_PER_PIXEL), 1.f);
    }
}

OPTIX_MISS_PROGRAM(Miss)() {
    const MissProgData &self = owl::getProgramData<MissProgData>();
    vec3f dir = optixGetWorldRayDirection();
    dir = normalize(dir);

    // Convert to spherical coords
    const float theta = atan2f(dir.z, dir.x);
    const float phi = acosf(dir.y);

    // Convert to texture coordinates
    const float u = theta / (2.0f * M_PIf) + 0.5f;
    const float v = phi / M_PIf;

    // Retrieve environment map color
    const vec4f bg_color = tex2D<float4>(self.env_map, u, v);

    Trace::Record& prd = owl::getPRD<Trace::Record>();
    prd.out.scatter_event = Trace::ScatterEvent::RayMissed;
    prd.out.attenuation = vec3f(bg_color.x, bg_color.y, bg_color.z);
}
