#include "hip/hip_runtime.h"
#include "Trace.cuh"
#include "trace/Ray.hpp"
#include "geometry/Sphere.hpp"

#include <optix_device.h>

#define SAMPLES_PER_PIXEL 4
#define MAX_DEPTH 16

OPTIX_BOUNDS_PROGRAM(LambertianSpheres)(const void *geom_data, box3f &prim_bounds, const int prim_id) {
    Geometry::Sphere::bounds<Geometry::LambertianSpheresGeom>(geom_data, prim_bounds, prim_id);
}

OPTIX_INTERSECT_PROGRAM(LambertianSpheres)() {
    Geometry::Sphere::intersect<Geometry::LambertianSpheresGeom>();
}

OPTIX_CLOSEST_HIT_PROGRAM(LambertianSpheres)() {
    Geometry::Sphere::closest_hit<Geometry::LambertianSpheresGeom, Material::Lambertian>();
}

inline __device__
vec3f miss_color(const Ray& ray) {
    const vec3f ray_dir = normalize(ray.direction);
    const float t = 0.5f * (ray_dir.y + 1.0f);
    const vec3f c = (1.0f - t) * vec3f(1.0f) + t * vec3f(0.5f, 0.7f, 1.0f);
    return c;
}

inline __device__
int sample_env_discrete(const RayGenData& self, Trace::Record& prd) {
    const int size = self.env.size.x * self.env.size.y;
    const float float_idx = prd.random() * size;
    const int idx = static_cast<float>(float_idx);
    float remainder = float_idx - idx;

    if (remainder < self.env.alias_pdf[idx]) {
        prd.out.pdf = self.env.pdf[idx];
        return idx;
    } else {
        const int alias = self.env.alias_i[idx];
        prd.out.pdf = self.env.pdf[alias];
        return alias;
    }
}

inline __device__
vec3f sample_env_ray(const RayGenData& self, Trace::Record& prd) {
    int idx = sample_env_discrete(self, prd);

    int y = idx / self.env.size.x;
    int x = idx % self.env.size.x;

    // Convert to 2d texture coordinates
    const float u = (x + 0.5) / self.env.size.x;
    const float v = (y + 0.5) / self.env.size.y;

    // Convert to spherical
    const float theta = 2.0f * M_PIf * (u - 0.5f);
    const float phi = M_PIf * v;

    // Convert to cartesian
    const float ray_x = cosf(theta) * sinf(phi);
    const float ray_y = cosf(phi);
    const float ray_z = sinf(theta) * sinf(phi);

    return vec3f(ray_x, ray_y, ray_z);
}

inline __device__
float get_env_pdf(const RayGenData& self, const vec3f& dir, Trace::Record& prd) {
    // Convert to spherical coords
    const float theta = atan2f(dir.z, dir.x);
    const float phi = acosf(dir.y);

    // Convert to texture coordinates
    const float u = theta / (2.0f * M_PIf) + 0.5f;
    const float v = phi / M_PIf;

    // printf("u: %f, v: %f\n", u, v);

    // Convert to index in pdf table
    const int x = static_cast<int>(u * self.env.size.x);
    const int y = static_cast<int>(v * self.env.size.y);
    const int idx = y * self.env.size.x + x;

    // Get pdf
    return self.env.pdf[idx];
}

inline __device__
vec3f trace_path(const RayGenData& self, Ray& ray, Trace::Record& prd) {
    vec3f accum_attenuation = 1.f;

    for (int depth = 0; depth < MAX_DEPTH; depth++) {
        traceRay(self.world, ray, prd);

        if (prd.out.scatter_event == Trace::ScatterEvent::RayMissed) {
            // Missed the scene, return background color
            return accum_attenuation * prd.out.attenuation;
        }
        else if (prd.out.scatter_event == Trace::ScatterEvent::RayCancelled) {
            // Hit light source
            return vec3f(0.f);
        }
        else {
            const vec3f brdf = prd.out.attenuation;
            const vec3f dir = prd.out.scattered_direction;
            const float pdf = (dot(normalize(prd.out.normal), normalize(dir)) / M_PIf);

            // TODO: I want this to be the pdf of the env map
            // const vec3f env_dir = sample_env_ray(self,prd);
            // const float env_pdf = get_env_pdf(self, dir, prd);
            // printf("pdf: %f %f\n", prd.out.pdf, env_pdf);

            const vec3f throughput = brdf / pdf;
            float roulette_weight =
                1.0f - clamp(max(max(throughput.x, throughput.y), throughput.z), 0.0f, 1.0f);
            if (depth <= 3) roulette_weight = 0.f;

            if (prd.random() < roulette_weight) {
                return vec3f(0.f);
            }

            accum_attenuation *= (throughput / (1.0f - roulette_weight));

            ray = Ray(
                prd.out.scattered_origin,
                dir,
                1e-3f,
                1e10f
            );
        }
    }

    return vec3f(0.f);
}

OPTIX_RAYGEN_PROGRAM(RayGen)() {
    const RayGenData& self = owl::getProgramData<RayGenData>();
    // Get our pixel indices
    const vec2i pixel_id = owl::getLaunchIndex();
    const int pboOfs = pixel_id.x + self.pbo_size.x * pixel_id.y;

    // Build primary rays
    Trace::Record prd;
    prd.random.init(pboOfs, self.launch->frame.id);

    vec3f color = 0.f;
    for (int sample_id = 0; sample_id < SAMPLES_PER_PIXEL; sample_id++) {
        // Build primary ray
        Ray ray;

        const vec2f pixel_offset(prd.random(), prd.random());
        const vec2f uv = (vec2f(pixel_id) + pixel_offset) / vec2f(self.pbo_size);
        const vec3f origin = self.launch->camera.pos;
        const vec3f direction = self.launch->camera.dir_00
            + uv.x * self.launch->camera.dir_du
            + uv.y * self.launch->camera.dir_dv;

        ray.origin = origin;
        ray.direction = normalize(direction);

        // Trace
        prd.out.pdf = 1.f;
        color += trace_path(self, ray, prd);
    }

    if (self.launch->dirty) {
        self.pbo_ptr[pboOfs] = vec4f(color * (1.f / SAMPLES_PER_PIXEL), 1.f);
    } else {
        self.pbo_ptr[pboOfs] += vec4f(color * (1.f / SAMPLES_PER_PIXEL), 1.f);
    }
}

// OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)() {
//     vec3f& prd = owl::getPRD<vec3f>();
//
//     const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
//
//     // Compute normal
//     const int prim_id = optixGetPrimitiveIndex();
//     const vec3i index = self.index[prim_id];
//     const vec3f v0 = self.vertex[index.x];
//     const vec3f v1 = self.vertex[index.y];
//     const vec3f v2 = self.vertex[index.z];
//     const vec3f n = normalize(cross(v1 - v0, v2 - v0));
//
//     const vec3f ray_dir = optixGetWorldRayDirection();
// }

OPTIX_MISS_PROGRAM(Miss)() {
    const MissProgData &self = owl::getProgramData<MissProgData>();
    vec3f dir = optixGetWorldRayDirection();
    dir = normalize(dir);

    // Convert to spherical coords
    const float theta = atan2f(dir.z, dir.x);
    const float phi = acosf(dir.y);

    // Convert to texture coordinates
    const float u = theta / (2.0f * M_PIf) + 0.5f;
    const float v = phi / M_PIf;

    // Retrieve environment map color
    const vec4f bg_color = tex2D<float4>(self.env_map, u, v);

    Trace::Record& prd = owl::getPRD<Trace::Record>();
    prd.out.scatter_event = Trace::ScatterEvent::RayMissed;
    prd.out.attenuation = vec3f(bg_color.x, bg_color.y, bg_color.z);
}
